#include "hip/hip_runtime.h"
#include <cmath>
#include <iomanip>
#include <iostream>
#include <vector>
using namespace std;

#include "base/PDAG2.h"
#include "structure_learning/constants.h"
#include "structure_learning/gpuPC3.h"
#include "structure_learning/orientation.h"

namespace cuda3 {
#include "structure_learning/utils.cuh"

__global__ void calc_regret(double *regret) {
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n == 0) return;
  double sum = 1, a = 1;
  int bound = ceil(2 + sqrt(20 * n * log(10.0)));
  for (int k = 1; k <= bound; k++) {
    a *= (n - k + 1) / static_cast<double>(n);
    sum += a;
  }
  regret[n * max_dim] = 1;
  regret[n * max_dim + 1] = sum;
  for (int k = 3; k <= max_dim; k++) {
    regret[n * max_dim + k - 1] =
        regret[n * max_dim + k - 2] +
        n * regret[n * max_dim + k - 3] / static_cast<double>(k - 2);
  }
  for (int k = 0; k < max_dim; k++) {
    regret[n * max_dim + k] = log2(max(1.0, regret[n * max_dim + k]));
  }
}

__device__ bool ci_test_sc_level_0(int n_data, int n_i, int n_j,
                                   int *contingency_matrix, int *marginals_i,
                                   int *marginals_j, double *regret) {
  double mi = 0;
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      if (!contingency_matrix[k * n_j + l]) continue;
      mi += static_cast<double>(contingency_matrix[k * n_j + l]) / n_data *
            log2(static_cast<double>(n_data) * contingency_matrix[k * n_j + l] /
                 (static_cast<double>(marginals_i[k]) * marginals_j[l]));
    }
  }
  // R(X_i)
  double r_i = regret[n_data * max_dim + n_i - 1];
  // R(X_i|X_j)
  double r_ij = 0;
  for (int l = 0; l < n_j; l++) {
    r_ij += regret[marginals_j[l] * max_dim + n_i - 1];
  }
  // R(X_j)
  double r_j = regret[n_data * max_dim + n_j - 1];
  double r_ji = 0;
  for (int l = 0; l < n_i; l++) {
    r_ji += regret[marginals_i[l] * max_dim + n_j - 1];
  }
  double threshold = min(r_ij - r_i, r_ji - r_j) / n_data;
  // printf("threshold: %.7lf\n", threshold);
  return mi <= threshold;
}

__device__ bool ci_test_g2_level_0(int n_data, int n_i, int n_j,
                                   int *contingency_matrix, int *marginals_i,
                                   int *marginals_j) {
  if (n_i == 1 || n_j == 1) {
    return true;
  }
  double g2 = 0;
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      double expected =
          static_cast<double>(marginals_i[k]) * marginals_j[l] / n_data;
      double observed = contingency_matrix[k * n_j + l];
      if (observed != 0) {
        g2 += 2 * observed * log(observed / expected);
      }
    }
  }
  double pval = pchisq(g2, (n_i - 1) * (n_j - 1));
  return pval >= 0.05;
}

__global__ void PC_level_0(int citest_type, int n_node, int n_data,
                           uint8_t *data, int *G, int *n_states, double *regret,
                           int *model, int *stats) {
  int i = blockIdx.x;
  int j = blockIdx.y;
  if (i >= j) {
    return;
  }
  __shared__ int contingency_matrix[max_dim * max_dim];
  int n_i = n_states[i];
  int n_j = n_states[j];
  for (int k = threadIdx.x; k < n_i * n_j; k += blockDim.x) {
    contingency_matrix[k] = 0;
  }
  if (threadIdx.x == 0) {
    uint smid;
    asm volatile("mov.u32 %0, %smid;" : "=r"(smid));
    atomicAdd(stats + smid, 1);
    atomicAdd(stats + smid + sm_num, 1);
  }
  __syncthreads();
  for (int k = threadIdx.x; k < n_data; k += blockDim.x) {
    int idx = data[i * n_data + k] * n_j + data[j * n_data + k];
    atomicAdd(contingency_matrix + idx, 1);
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int marginals_i[max_dim];
    int marginals_j[max_dim];
    for (int k = 0; k < max_dim; k++) {
      marginals_i[k] = 0;
      marginals_j[k] = 0;
    }
    for (int k = 0; k < n_i; k++) {
      for (int l = 0; l < n_j; l++) {
        int entry = contingency_matrix[k * n_j + l];
        marginals_i[k] += entry;
        marginals_j[l] += entry;
      }
    }
    bool result;
    if (citest_type == 0) {
      result = ci_test_g2_level_0(n_data, n_i, n_j, contingency_matrix,
                                  marginals_i, marginals_j);
    } else {
      result = ci_test_sc_level_0(n_data, n_i, n_j, contingency_matrix,
                                  marginals_i, marginals_j, regret);
    }
    if (result) {
      G[i * n_node + j] = 0;
      G[j * n_node + i] = 0;
    }
    // bool sep_result = d_separated(0, n_node, i, j, nullptr, model);
    // if (result && sep_result) {
    //   atomicAdd(stats, 1);
    // } else if (result && !sep_result) {
    //   atomicAdd(stats + 1, 1);
    // } else if (!result && sep_result) {
    //   atomicAdd(stats + 2, 1);
    // } else {
    //   atomicAdd(stats + 3, 1);
    // }
  }
}

__device__ void ci_test_sc_level_n(double *mi, int n_data, int dim_s,
                                   int dim_mul, int n_i, int n_j, int *N_i_j_s,
                                   int *N_i_s, int *N_j_s, int *N_s,
                                   bool *result, double *regret) {
  double *r_i = mi + 1;
  double *r_ij = mi + 2;
  double *r_j = mi + 3;
  double *r_ji = mi + 4;
  if (threadIdx.x == 0) {
    *mi = *r_i = *r_ij = *r_j = *r_ji = 0;
  }
  __syncthreads();
  for (int h = threadIdx.x; h < dim_s; h += blockDim.x) {
    if (N_s[h] == 0) continue;
    atomicAdd(r_i, regret[N_s[h] * max_dim + n_i - 1]);
    atomicAdd(r_j, regret[N_s[h] * max_dim + n_j - 1]);
    for (int k = 0; k < n_i; k++) {
      atomicAdd(r_ji, regret[N_i_s[h * n_i + k] * max_dim + n_j - 1]);
      if (k && !N_i_s[h * n_i + k]) continue;
      for (int l = 0; l < n_j; l++) {
        if (k == 0) {
          atomicAdd(r_ij, regret[N_j_s[h * n_j + l] * max_dim + n_i - 1]);
        }
        int g = (h / dim_mul) * dim_mul * n_j + l * dim_mul + h % dim_mul;
        double entry = N_i_j_s[g * n_i + k];
        if (entry) {
          atomicAdd(mi, entry / n_data *
                            log2(N_s[h] * entry /
                                 (static_cast<double>(N_i_s[h * n_i + k]) *
                                  N_j_s[h * n_j + l])));
        }
      }
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    double threshold = min(*r_ij - *r_i, *r_ji - *r_j) / n_data;
    *result = (*mi <= threshold);
  }
}

__device__ void ci_test_sc_level_n_2(double *mi, int n_data, int dim_s,
                                     int dim_mul_i, int dim_mul_j, int n_i,
                                     int n_j, int *N_i_j_s, int *N_i_s,
                                     int *N_j_s, int *N_s, bool *result,
                                     double *regret) {
  double *r_i = mi + 1;
  double *r_ij = mi + 2;
  double *r_j = mi + 3;
  double *r_ji = mi + 4;
  if (threadIdx.x == 0) {
    *mi = *r_i = *r_ij = *r_j = *r_ji = 0;
  }
  __syncthreads();
  for (int h = threadIdx.x; h < dim_s; h += blockDim.x) {
    if (N_s[h] == 0) continue;
    atomicAdd(r_i, regret[N_s[h] * max_dim + n_i - 1]);
    atomicAdd(r_j, regret[N_s[h] * max_dim + n_j - 1]);
    for (int k = 0; k < n_i; k++) {
      atomicAdd(r_ji, regret[N_i_s[h * n_i + k] * max_dim + n_j - 1]);
      for (int l = 0; l < n_j; l++) {
        if (k == 0) {
          atomicAdd(r_ij, regret[N_j_s[h * n_j + l] * max_dim + n_i - 1]);
        }
        int g = (h / (dim_mul_j / n_i)) * dim_mul_j * n_j + l * dim_mul_j +
                (h % (dim_mul_j / n_i)) / dim_mul_i * dim_mul_i * n_i +
                k * dim_mul_i + h % dim_mul_i;
        double entry = N_i_j_s[g];
        if (entry) {
          atomicAdd(mi, entry / n_data *
                            log2(N_s[h] * entry /
                                 (static_cast<double>(N_i_s[h * n_i + k]) *
                                  N_j_s[h * n_j + l])));
        }
      }
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    double threshold = min(*r_ij - *r_i, *r_ji - *r_j) / n_data;
    *result = (*mi <= threshold);
  }
}

__device__ void ci_test_g2_level_n(double *g2, int n_data, int dim_s,
                                   int dim_mul, int n_i, int n_j, int *N_i_j_s,
                                   int *N_i_s, int *N_j_s, int *N_s,
                                   bool *result) {
  int *df = reinterpret_cast<int *>(g2 + 1);
  if (threadIdx.x == 0) {
    *g2 = 0;
    *df = 0;
  }
  __syncthreads();
  for (int h = threadIdx.x; h < dim_s; h += blockDim.x) {
    if (N_s[h] == 0) continue;
    int alx = 0, aly = 0;
    for (int l = 0; l < n_j; l++) {
      aly += (N_j_s[h * n_j + l] > 0);
    }
    for (int k = 0; k < n_i; k++) {
      if (!N_i_s[h * n_i + k]) continue;
      alx++;
      for (int l = 0; l < n_j; l++) {
        int g = (h / dim_mul) * dim_mul * n_j + l * dim_mul + h % dim_mul;
        double expected = static_cast<double>(N_i_s[h * n_i + k]) *
                          N_j_s[h * n_j + l] / N_s[h];
        double observed = N_i_j_s[g * n_i + k];
        if (observed) {
          double sum_term = 2 * observed * log(observed / expected);
          atomicAdd(g2, sum_term);
        }
      }
    }
    alx = (alx >= 1 ? alx : 1);
    aly = (aly >= 1 ? aly : 1);
    atomicAdd(df, (alx - 1) * (aly - 1));
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    if (df == 0) {
      *result = true;
    } else {
      double pval = pchisq(*g2, *df);
      *result = (pval >= 0.05);
    }
  }
}

__device__ void ci_test_g2_level_n_2(double *g2, int n_data, int dim_s,
                                     int dim_mul_i, int dim_mul_j, int n_i,
                                     int n_j, int *N_i_j_s, int *N_i_s,
                                     int *N_j_s, int *N_s, bool *result) {
  int *df = reinterpret_cast<int *>(g2 + 1);
  if (threadIdx.x == 0) {
    *g2 = 0;
    *df = 0;
  }
  __syncthreads();
  for (int h = threadIdx.x; h < dim_s; h += blockDim.x) {
    if (N_s[h] == 0) continue;
    int alx = 0, aly = 0;
    for (int l = 0; l < n_j; l++) {
      aly += (N_j_s[h * n_j + l] > 0);
    }
    for (int k = 0; k < n_i; k++) {
      if (!N_i_s[h * n_i + k]) continue;
      alx++;
      for (int l = 0; l < n_j; l++) {
        int g = (h / (dim_mul_j / n_i)) * dim_mul_j * n_j + l * dim_mul_j +
                (h % (dim_mul_j / n_i)) / dim_mul_i * dim_mul_i * n_i +
                k * dim_mul_i + h % dim_mul_i;
        double expected = static_cast<double>(N_i_s[h * n_i + k]) *
                          N_j_s[h * n_j + l] / N_s[h];
        double observed = N_i_j_s[g];
        if (observed) {
          double sum_term = 2 * observed * log(observed / expected);
          atomicAdd(g2, sum_term);
        }
      }
    }
    alx = (alx >= 1 ? alx : 1);
    aly = (aly >= 1 ? aly : 1);
    atomicAdd(df, (alx - 1) * (aly - 1));
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    if (df == 0) {
      *result = true;
    } else {
      double pval = pchisq(*g2, *df);
      *result = (pval >= 0.05);
    }
  }
}

__global__ void PC_level_n(int citest_type, int level, int n_node, int n_data,
                           uint8_t *data, int *G, int *n_states,
                           bool use_working_memory, int *working_memory,
                           int *sepsets, double *regret, int *model,
                           int *stats) {
  extern __shared__ int smem[];
  for (int i = blockIdx.x; i < n_node; i += gridDim.x) {
    __syncthreads();
    int *G_compacted = smem;
    if (threadIdx.x == 0) {
      int cnt = 0;
      for (int j = 0; j < n_node; j++) {
        if (G[i * n_node + j]) {
          G_compacted[++cnt] = j;
        }
      }
      G_compacted[0] = cnt;
    }
    __syncthreads();
    int n_adj = G_compacted[0];
    if (n_adj - 1 < level) {
      continue;
    }
    int max_dim_s = pow(static_cast<double>(max_dim), level);
    int reserved_size_per_ci_test =
        max_dim_s * max_dim * max_dim + 2 * max_dim_s * max_dim + max_dim_s;
    int sepset_cnt = binom(n_adj, level + 1);
    int *sepset = smem + n_adj + 2;
    int *dim_mul = smem + n_adj + 2 + level + 1;
    int *connected_to_all = smem + n_adj + 2 + (level + 1) * 2;
    int *thread_memory;
    if (use_working_memory) {
      thread_memory = working_memory + (gridDim.y * blockIdx.x + blockIdx.y) *
                                           reserved_size_per_ci_test;
    } else {
      thread_memory = smem + n_adj + 2 + (level + 1) * 3;
    }
    int n_i = n_states[i];
    for (int sepset_idx = blockIdx.y; sepset_idx < sepset_cnt;
         sepset_idx += gridDim.y) {
      __syncthreads();
      int *valid = smem + n_adj + 1;
      if (threadIdx.x == 0) {
        comb(n_adj, level + 1, sepset_idx, -1, sepset);
        *valid = 0;
        dim_mul[0] = 1;
        for (int k = 0; k < level + 1; k++) {
          sepset[k] = G_compacted[sepset[k] + 1];
          if (G[i * n_node + sepset[k]] == 1) {
            *valid = 1;
          }
          if (k < level) {
            dim_mul[k + 1] = dim_mul[k] * n_states[sepset[k]];
          }
        }
        for (int idx_j = 0; idx_j < level + 1; idx_j++) {
          int j = sepset[idx_j];
          connected_to_all[idx_j] = 1;
          bool exist_nondeleted_edge = false;
          for (int idx_k = 0; idx_k < level + 1; idx_k++) {
            if (idx_j != idx_k && !G[j * n_node + sepset[idx_k]]) {
              connected_to_all[idx_j] = 0;
              break;
            }
            if (idx_j != idx_k && G[j * n_node + sepset[idx_k]] == 1) {
              exist_nondeleted_edge = true;
            }
          }
          if (connected_to_all[idx_j]) {
            if (j < i) {
              *valid = 0;
              break;
            }
            if (j > i && exist_nondeleted_edge) {
              *valid = 1;
            }
          }
        }
      }
      __syncthreads();
      if (*valid == 0) continue;
      if (threadIdx.x == 0) {
        uint smid;
        asm volatile("mov.u32 %0, %smid;" : "=r"(smid));
        atomicAdd(stats + smid + sm_num, 1);
      }
      int dim_s = dim_mul[level] * n_states[sepset[level]];
      int *N_i_j_s = thread_memory;
      if (threadIdx.x == 0) {
        int malloc_size = dim_s * n_i;
        memset(N_i_j_s, 0, malloc_size * sizeof(int));
      }
      __syncthreads();
      for (int k = threadIdx.x; k < n_data; k += blockDim.x) {
        int val_i = data[i * n_data + k];
        int s_idx = 0;
        for (int l = 0; l < level + 1; l++) {
          s_idx += data[sepset[l] * n_data + k] * dim_mul[l];
        }
        atomicAdd(N_i_j_s + s_idx * n_i + val_i, 1);
      }
      int scratch_addr = n_adj + 2 + (level + 1) * 3 +
                         (use_working_memory ? 0 : reserved_size_per_ci_test);
      scratch_addr = (scratch_addr + 1) / 2 * 2;
      double *scratch_ptr = reinterpret_cast<double *>(smem + scratch_addr);
      bool result;
      for (int idx_j = 0; idx_j < level; idx_j++) {
        int j = sepset[idx_j];
        for (int idx_k = idx_j + 1; idx_k < level + 1; idx_k++) {
          int k = sepset[idx_k];
          if (threadIdx.x == 0) {
            if (G[j * n_node + k] != 1) {
              *valid = 0;
            } else {
              *valid = (connected_to_all[idx_j] || connected_to_all[idx_k]);
            }
          }
          __syncthreads();
          if (*valid == 0) continue;
          if (threadIdx.x == 0) {
            uint smid;
            asm volatile("mov.u32 %0, %smid;" : "=r"(smid));
            atomicAdd(stats + smid, 1);
          }
          int n_j = n_states[j];
          int n_k = n_states[k];
          int *N_i_s = N_i_j_s + dim_s * n_i;
          int *N_j_s = N_i_s + dim_s * n_i / n_k;
          int *N_s = N_j_s + dim_s * n_i / n_j;
          if (threadIdx.x == 0) {
            int malloc_size =
                dim_s * n_i / n_k + dim_s * n_i / n_j + dim_s * n_i / n_j / n_k;
            memset(N_i_s, 0, malloc_size * sizeof(int));
          }
          __syncthreads();
          for (int g = threadIdx.x; g < dim_s * n_i; g += blockDim.x) {
            int h =
                g / (dim_mul[idx_k] * n_i) / n_k * dim_mul[idx_k] * n_i / n_j +
                g % (dim_mul[idx_k] * n_i) / (dim_mul[idx_j] * n_i) / n_j *
                    dim_mul[idx_j] * n_i +
                g % (dim_mul[idx_j] * n_i);
            int k = g / (dim_mul[idx_j] * n_i) % n_j;
            int l = g / (dim_mul[idx_k] * n_i) % n_k;
            int entry = N_i_j_s[g];
            atomicAdd(N_i_s + h * n_j + k, entry);
            atomicAdd(N_j_s + h * n_k + l, entry);
            atomicAdd(N_s + h, entry);
          }
          if (citest_type == 0) {
            ci_test_g2_level_n_2(scratch_ptr, n_data, dim_s / n_j / n_k * n_i,
                                 dim_mul[idx_j] * n_i, dim_mul[idx_k] * n_i,
                                 n_j, n_k, N_i_j_s, N_i_s, N_j_s, N_s, &result);
          } else {
            ci_test_sc_level_n_2(scratch_ptr, n_data, dim_s / n_j / n_k * n_i,
                                 dim_mul[idx_j] * n_i, dim_mul[idx_k] * n_i,
                                 n_j, n_k, N_i_j_s, N_i_s, N_j_s, N_s, &result,
                                 regret);
          }
          if (threadIdx.x == 0 && result) {
            if (atomicCAS(G + j * n_node + k, 1, -1) == 1) {
              G[k * n_node + j] = -1;
              sepsets[(j * n_node + k) * max_level] = i;
              int p = 1;
              for (int l = 0; l < level + 1; l++) {
                if (l == idx_j || l == idx_k) continue;
                sepsets[(j * n_node + k) * max_level + p] = sepset[l];
                p++;
              }
            }
          }
        }
      }
      for (int idx_j = 0; idx_j < level + 1; idx_j++) {
        __syncthreads();
        int j = sepset[idx_j];
        if (threadIdx.x == 0) {
          *valid = (G[i * n_node + j] == 1);
        }
        __syncthreads();
        if (*valid == 0) continue;
        if (threadIdx.x == 0) {
          uint smid;
          asm volatile("mov.u32 %0, %smid;" : "=r"(smid));
          atomicAdd(stats + smid, 1);
        }
        int n_j = n_states[j];
        int *N_i_s = N_i_j_s + dim_s * n_i;
        int *N_j_s = N_i_s + dim_s * n_i / n_j;
        int *N_s = N_j_s + dim_s;
        if (threadIdx.x == 0) {
          int malloc_size = dim_s * n_i / n_j + dim_s + dim_s / n_j;
          memset(N_i_s, 0, malloc_size * sizeof(int));
        }
        __syncthreads();
        for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
          for (int k = 0; k < n_i; k++) {
            int h = (g / dim_mul[idx_j]) / n_j * dim_mul[idx_j] +
                    g % dim_mul[idx_j];
            int l = g / dim_mul[idx_j] % n_j;
            int entry = N_i_j_s[g * n_i + k];
            atomicAdd(N_i_s + h * n_i + k, entry);
            atomicAdd(N_j_s + h * n_j + l, entry);
            atomicAdd(N_s + h, entry);
          }
        }
        if (citest_type == 0) {
          ci_test_g2_level_n(scratch_ptr, n_data, dim_s / n_j, dim_mul[idx_j],
                             n_i, n_j, N_i_j_s, N_i_s, N_j_s, N_s, &result);
        } else {
          ci_test_sc_level_n(scratch_ptr, n_data, dim_s / n_j, dim_mul[idx_j],
                             n_i, n_j, N_i_j_s, N_i_s, N_j_s, N_s, &result,
                             regret);
        }
        if (threadIdx.x == 0 && result) {
          int ij_min = (i < j ? i : j);
          int ij_max = (i < j ? j : i);
          if (atomicCAS(G + ij_min * n_node + ij_max, 1, -1) == 1) {
            G[ij_max * n_node + ij_min] = -1;
            int p = 0;
            for (int k = 0; k < level + 1; k++) {
              if (k == idx_j) continue;
              sepsets[(ij_min * n_node + ij_max) * max_level + p] = sepset[k];
              p++;
            }
          }
        }
        // if (threadIdx.x == 0) {
        //   int sepset2[max_level];
        //   int p = 0;
        //   for (int k = 0; k < level + 1; k++) {
        //     if (k == idx_j) continue;
        //     sepset2[p] = sepset[k];
        //     p++;
        //   }
        //   bool sep_result = d_separated(level, n_node, i, j, sepset2, model);
        //   if (result && sep_result) {
        //     atomicAdd(stats, 1);
        //   } else if (result && !sep_result) {
        //     atomicAdd(stats + 1, 1);
        //   } else if (!result && sep_result) {
        //     atomicAdd(stats + 2, 1);
        //   } else {
        //     atomicAdd(stats + 3, 1);
        //   }
        // }
        __syncthreads();
      }
    }
  }
}

PDAG PCsearch(int citest_type, int n_node, int n_data,
              const vector<uint8_t> &data, const vector<int> &n_states,
              const vector<int> &model) {
  vector<int> G(n_node * n_node);
  for (int i = 0; i < n_node; i++) {
    for (int j = 0; j < n_node; j++) {
      if (i != j) G[i * n_node + j] = 1;
    }
  }
  vector<int> sepsets(n_node * n_node * max_level, -1);
  uint8_t *data_d;
  vector<double> regret(n_data * max_dim * 2);
  vector<int> stats(sm_num * 2);
  int *G_d, *n_states_d, *working_memory_d, *sepsets_d, *model_d, *stats_d;
  double *regret_d;
  int size_G = sizeof(int) * n_node * n_node;
  int size_data = sizeof(uint8_t) * n_data * n_node;
  int size_n_states = sizeof(int) * n_node;
  int size_working_memory = sizeof(int) * 500'000'000;
  int size_sepsets = sizeof(int) * n_node * n_node * max_level;
  int size_regret = sizeof(double) * n_data * max_dim * 2;
  int size_model = sizeof(int) * n_node * n_node * 2;
  int size_stats = sizeof(int) * sm_num * 2;
  CUDA_CHECK(hipMalloc(&G_d, size_G));
  CUDA_CHECK(hipMalloc(&data_d, size_data));
  CUDA_CHECK(hipMalloc(&n_states_d, size_n_states));
  CUDA_CHECK(hipMalloc(&working_memory_d, size_working_memory));
  CUDA_CHECK(hipMalloc(&sepsets_d, size_sepsets));
  CUDA_CHECK(hipMalloc(&regret_d, size_regret));
  CUDA_CHECK(hipMalloc(&model_d, size_model));
  CUDA_CHECK(hipMalloc(&stats_d, size_stats));
  CUDA_CHECK(
      hipMemcpy(data_d, data.data(), size_data, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(n_states_d, n_states.data(), size_n_states,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(sepsets_d, sepsets.data(), size_sepsets,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(model_d, model.data(), size_model, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(stats_d, stats.data(), size_stats, hipMemcpyHostToDevice));

  calc_regret<<<n_data / 1024 + 1, 1024>>>(regret_d);
  CUDA_CHECK(
      hipMemcpy(regret.data(), regret_d, size_regret, hipMemcpyDeviceToHost));

  int true_max_deg = 0, true_max_indeg = 0, true_max_outdeg = 0;
  for (size_t i = 0; i < n_node; i++) {
    true_max_deg =
        max(true_max_deg,
            model.at(i * n_node) + model.at(n_node * n_node + i * n_node));
    true_max_indeg =
        max(true_max_indeg, model.at(n_node * n_node + i * n_node));
    true_max_outdeg = max(true_max_outdeg, model.at(i * n_node));
  }
  cout << "true_max_deg: " << true_max_deg
       << ", true_max_indeg: " << true_max_indeg
       << ", true_max_outdeg: " << true_max_outdeg << endl;
  // cout << "---regret---" << endl;
  // for (int n = n_data - 10; n <= n_data; n++) {
  //   for (int k = 1; k <= max_dim; k++) {
  //     cout << regret[n * max_dim + k - 1] << " ";
  //   }
  //   cout << endl;
  // }
  // cout << "---" << endl;
  vector<float> buf1;
  vector<int> buf2;
  hipEvent_t start, stop;
  CUDA_CHECK(hipEventCreate(&start));
  CUDA_CHECK(hipEventCreate(&stop));
  // stage 1: Do CI tests between nodes and remove edge (undirected graph)
  int level = 0;
  int max_n_adj = n_node - 1;
  uint64_t max_dim_s = 1;
  while (level <= n_node - 2) {
    CUDA_CHECK(hipMemcpy(G_d, G.data(), size_G, hipMemcpyHostToDevice));
    stats = vector<int>(sm_num * 2);
    CUDA_CHECK(
        hipMemcpy(stats_d, stats.data(), size_stats, hipMemcpyHostToDevice));
    cout << "level: " << level << ", max_n_adj: " << max_n_adj << endl;
    if (level > max_level) break;
    CUDA_CHECK(hipEventRecord(start));
    if (level == 0) {
      dim3 threadsPerBlock(64);
      dim3 numBlocks(n_node, n_node);
      PC_level_0<<<numBlocks, threadsPerBlock>>>(citest_type, n_node, n_data,
                                                 data_d, G_d, n_states_d,
                                                 regret_d, model_d, stats_d);
    } else {
      dim3 threadsPerBlock(64);
      dim3 numBlocks(n_node, max_n_adj * 2);
      uint64_t reserved_size_per_ci_test =
          max_dim_s * max_dim * max_dim + 2 * max_dim_s * max_dim + max_dim_s;
      if (reserved_size_per_ci_test > size_working_memory / sizeof(int)) {
        cout << "working memory is not enough" << endl;
        break;
      }
      if (reserved_size_per_ci_test * 2 < 1000) {
        PC_level_n<<<numBlocks, threadsPerBlock,
                     sizeof(int) * (max_n_adj + 2 + (level + 1) * 3 +
                                    reserved_size_per_ci_test) +
                         sizeof(double) * (5 + 1)>>>(
            citest_type, level, n_node, n_data, data_d, G_d, n_states_d, false,
            nullptr, sepsets_d, regret_d, model_d, stats_d);
      } else {
        uint64_t reserved_size_per_row =
            reserved_size_per_ci_test * numBlocks.y;
        int max_rows =
            size_working_memory / sizeof(int) / reserved_size_per_row;
        if (max_rows == 0) {
          int max_columns =
              size_working_memory / sizeof(int) / reserved_size_per_ci_test;
          numBlocks.x = 1;
          numBlocks.y = max_columns;
        } else if (numBlocks.x > max_rows) {
          numBlocks.x = max_rows;
        }
        cout << "numBlocks: " << numBlocks.x << ", " << numBlocks.y << endl;
        cout << "threadsPerBlock: " << threadsPerBlock.x << endl;
        PC_level_n<<<numBlocks, threadsPerBlock,
                     sizeof(int) * (max_n_adj + 2 + (level + 1) * 3) +
                         sizeof(double) * (5 + 1)>>>(
            citest_type, level, n_node, n_data, data_d, G_d, n_states_d, true,
            working_memory_d, sepsets_d, regret_d, model_d, stats_d);
      }
    }
    CUDA_CHECK(
        hipMemcpy(stats.data(), stats_d, size_stats, hipMemcpyDeviceToHost));
    CUDA_CHECK(hipEventRecord(stop));
    CUDA_CHECK(hipEventSynchronize(stop));
    float milliseconds = 0;
    CUDA_CHECK(hipEventElapsedTime(&milliseconds, start, stop));
    buf1.push_back(milliseconds / 1000);
    // cout << "----stats: " << endl;
    // for (int i = 0; i < sm_num; i++) {
    //   cout << i << ", " << stats[i] << ", " << stats[i + sm_num] << endl;
    // }
    // cout << "-----" << endl;
    buf2.push_back(accumulate(stats.begin(), stats.begin() + sm_num, 0));
    buf2.push_back(
        accumulate(stats.begin() + sm_num, stats.begin() + sm_num * 2, 0));
    CUDA_CHECK(hipMemcpy(G.data(), G_d, size_G, hipMemcpyDeviceToHost));
    max_n_adj = 0;
    int next_node_cnt = 0;
    int next_edge_cnt = 0;
    for (int i = 0; i < n_node; i++) {
      int n_adj = 0;
      for (int j = 0; j < n_node; j++) {
        if (G[i * n_node + j] == -1) {
          G[i * n_node + j] = 0;
        }
        if (G[i * n_node + j]) n_adj++;
      }
      if (n_adj - 1 > level) {
        next_node_cnt++;
      }
      max_n_adj = max(max_n_adj, n_adj);
      next_edge_cnt += n_adj;
    }
    cout << "next_node_cnt, next_edge_cnt: " << next_node_cnt << ", "
         << next_edge_cnt / 2 << endl;
    if (max_n_adj - 1 <= level) break;
    level++;
    max_dim_s *= max_dim;
  }
  CUDA_CHECK(hipMemcpy(sepsets.data(), sepsets_d, size_sepsets,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(G_d));
  CUDA_CHECK(hipFree(data_d));
  CUDA_CHECK(hipFree(n_states_d));
  CUDA_CHECK(hipFree(working_memory_d));
  CUDA_CHECK(hipFree(sepsets_d));
  CUDA_CHECK(hipEventDestroy(start));
  CUDA_CHECK(hipEventDestroy(stop));
  for (float f : buf1) {
    cout << fixed << setprecision(3) << f << endl;
  }
  for (int c : buf2) {
    cout << c << endl;
  }
  // stage 2: orient edges
  PDAG G_pdag;
  G_pdag.g = vector<vector<bool>>(n_node, vector<bool>(n_node));
  for (int i = 0; i < n_node; i++) {
    for (int j = 0; j < n_node; j++) {
      G_pdag.g.at(i).at(j) = G[i * n_node + j];
    }
  }
  orientation(G_pdag, sepsets);
  return G_pdag;
}

py::array_t<bool> gpuPC3(int citest_type, py::array_t<uint8_t> data,
                         py::array_t<int> n_states,
                         py::array_t<bool> true_model) {
  // translate input data to c++ vector(this is not optimal but I don't know
  // how to use pybind11::array_t)
  py::buffer_info buf_data = data.request(), buf_states = n_states.request(),
                  buf_model = true_model.request();
  const uint8_t *__restrict__ prt_data = static_cast<uint8_t *>(buf_data.ptr);
  const int *__restrict__ prt_states = static_cast<int *>(buf_states.ptr);
  const bool *__restrict__ prt_model = static_cast<bool *>(buf_model.ptr);
  size_t n_data = buf_data.shape[0],
         n_node = buf_data.shape[1];  // number of nodes
  cout << "n_data, n_node: " << n_data << ' ' << n_node << endl;
  vector<uint8_t> data_vec(n_data * n_node);
  vector<int> n_states_vec(n_node);
  vector<int> model_vec(2 * n_node * n_node);
  for (size_t i = 0; i < n_data; i++) {
    for (size_t j = 0; j < n_node; j++) {
      data_vec.at(j * n_data + i) = prt_data[i * n_node + j];
    }
  }
  for (size_t i = 0; i < n_node; i++) {
    n_states_vec.at(i) = prt_states[i];
  }
  for (size_t i = 0; i < n_node; i++) {
    int cnt = 0;
    for (size_t j = 0; j < n_node; j++) {
      if (prt_model[i * n_node + j]) {
        model_vec.at(i * n_node + (++cnt)) = j;
      }
    }
    model_vec.at(i * n_node) = cnt;
  }
  for (size_t i = 0; i < n_node; i++) {
    int cnt = 0;
    for (size_t j = 0; j < n_node; j++) {
      if (prt_model[j * n_node + i]) {
        model_vec.at(n_node * n_node + i * n_node + (++cnt)) = j;
      }
    }
    model_vec.at(n_node * n_node + i * n_node) = cnt;
  }
  auto endg = py::array_t<bool>({n_node, n_node});
  py::buffer_info buf_endg = endg.request();
  bool *__restrict__ prt_endg = static_cast<bool *>(buf_endg.ptr);

  PDAG Gend =
      PCsearch(citest_type, n_node, n_data, data_vec, n_states_vec, model_vec);

  // translate Gend to py::array_t (this is not optimal but I don't know how
  // to use pybind11::array_t)
  for (size_t i = 0; i < n_node; i++) {
    for (size_t j = 0; j < n_node; j++) {
      prt_endg[i * n_node + j] = Gend.g.at(i).at(j);
    }
  }
  return endg;
}
}  // namespace cuda3