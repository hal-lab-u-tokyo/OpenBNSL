#include "hip/hip_runtime.h"
#include <cmath>
#include <iostream>
#include <vector>
using namespace std;

#include "base/PDAG2.h"
#include "structure_learning/constants.h"
#include "structure_learning/gpuPC.h"
#include "structure_learning/orientation.h"

namespace cuda {
#include "structure_learning/utils.cuh"

// Based on: "GPU-Accelerated Constraint-Based Causal Structure Learning for
// Discrete Data.", 2021
//   authors: Hagedorn, Christopher, and Johannes Huegle
//   journal: Proceedings of the 2021 SIAM International Conference on Data
//   Mining (SDM)

__global__ void calc_regret(double *regret) {
  int n = blockDim.x * blockIdx.x + threadIdx.x;
  if (n == 0) return;
  double sum = 1, a = 1;
  for (int k = 1; k <= n; k++) {
    a *= (n - k + 1) / static_cast<double>(n);
    sum += a;
  }
  regret[n * max_dim] = 1;
  regret[n * max_dim + 1] = sum;
  for (int k = 3; k <= max_dim; k++) {
    regret[n * max_dim + k - 1] =
        regret[n * max_dim + k - 2] +
        n * regret[n * max_dim + k - 3] / static_cast<double>(k - 2);
  }
}

__device__ bool ci_test_chi_squared_level_0(int n_data, int n_i, int n_j,
                                            int *contingency_matrix,
                                            int *marginals_i,
                                            int *marginals_j) {
  if (n_i == 1 || n_j == 1) {
    return true;
  }
  double chi_squared = 0;
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      double expected =
          static_cast<double>(marginals_i[k]) * marginals_j[l] / n_data;
      double observed = contingency_matrix[k * n_j + l];
      chi_squared += (observed - expected) * (observed - expected) / expected;
    }
  }
  double pval = pchisq(chi_squared, (n_i - 1) * (n_j - 1));
  return pval >= 0.01;
}

__device__ bool ci_test_mi_level_0(int n_data, int n_i, int n_j,
                                   int *contingency_matrix, int *marginals_i,
                                   int *marginals_j) {
  double mi = 0;
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      if (!contingency_matrix[k * n_j + l]) continue;
      mi += static_cast<double>(contingency_matrix[k * n_j + l]) / n_data *
            log2(static_cast<double>(n_data) * contingency_matrix[k * n_j + l] /
                 (static_cast<double>(marginals_i[k]) * marginals_j[l]));
    }
  }
  return mi < 0.003;
}

__device__ bool ci_test_sc_level_0(int n_data, int n_i, int n_j,
                                   int *contingency_matrix, int *marginals_i,
                                   int *marginals_j, double *regret) {
  double mi = 0;
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      if (!contingency_matrix[k * n_j + l]) continue;
      mi += static_cast<double>(contingency_matrix[k * n_j + l]) / n_data *
            log2(static_cast<double>(n_data) * contingency_matrix[k * n_j + l] /
                 (static_cast<double>(marginals_i[k]) * marginals_j[l]));
    }
  }
  // R(X_i)
  double r_i = log2(max(1.0, regret[n_data * max_dim + n_i - 1]));
  // R(X_i|X_j)
  double r_ij = 0;
  for (int l = 0; l < n_j; l++) {
    r_ij += log2(max(1.0, regret[marginals_j[l] * max_dim + n_i - 1]));
  }
  // R(X_j)
  double r_j = log2(max(1.0, regret[n_data * max_dim + n_j - 1]));
  double r_ji = 0;
  for (int l = 0; l < n_i; l++) {
    r_ji += log2(max(1.0, regret[marginals_i[l] * max_dim + n_j - 1]));
  }
  double threshold = min(r_ij - r_i, r_ji - r_j) / n_data;
  // printf("threshold: %.7lf\n", threshold);
  return mi <= threshold;
}

__device__ bool ci_test_g2_level_0(int n_data, int n_i, int n_j,
                                   int *contingency_matrix, int *marginals_i,
                                   int *marginals_j) {
  if (n_i == 1 || n_j == 1) {
    return true;
  }
  double g2 = 0;
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      double expected =
          static_cast<double>(marginals_i[k]) * marginals_j[l] / n_data;
      double observed = contingency_matrix[k * n_j + l];
      if (observed != 0) {
        g2 += 2 * observed * log(observed / expected);
      }
    }
  }
  double pval = pchisq(g2, (n_i - 1) * (n_j - 1));
  return pval >= 0.05;
}

__device__ bool ci_test_bayes_factor_level_0(int n_data, int n_i, int n_j,
                                             int *contingency_matrix,
                                             int *marginals_i,
                                             int *marginals_j) {
  double independent_score = 0;
  double dependent_score = 0;
  const double alpha = 0.5;
  for (int k = 0; k < n_i; k++) {
    independent_score += lgamma(marginals_i[k] + alpha) - lgamma(alpha);
  }
  independent_score += lgamma(n_i * alpha) - lgamma(n_i * alpha + n_data);
  for (int l = 0; l < n_j; l++) {
    independent_score += lgamma(marginals_j[l] + alpha) - lgamma(alpha);
  }
  independent_score += lgamma(n_j * alpha) - lgamma(n_j * alpha + n_data);
  for (int k = 0; k < n_i; k++) {
    for (int l = 0; l < n_j; l++) {
      dependent_score +=
          lgamma(contingency_matrix[k * n_j + l] + alpha) - lgamma(alpha);
    }
  }
  dependent_score +=
      lgamma(n_i * n_j * alpha) - lgamma(n_i * n_j * alpha + n_data);
  return independent_score > dependent_score - 1e-10;
}

__global__ void PC_level_0(int n_node, int n_data, uint8_t *data, int *G,
                           int *n_states, double *regret, int *model,
                           int *stats) {
  int i = blockIdx.x;
  int j = blockIdx.y;
  if (i >= j) {
    return;
  }
  __shared__ int contingency_matrix[max_dim * max_dim];
  int n_i = n_states[i];
  int n_j = n_states[j];
  for (int k = threadIdx.x; k < n_i * n_j; k += blockDim.x) {
    contingency_matrix[k] = 0;
  }
  __syncthreads();
  for (int k = threadIdx.x; k < n_data; k += blockDim.x) {
    int idx = data[i * n_data + k] * n_j + data[j * n_data + k];
    atomicAdd(contingency_matrix + idx, 1);
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    int marginals_i[max_dim];
    int marginals_j[max_dim];
    for (int k = 0; k < max_dim; k++) {
      marginals_i[k] = 0;
      marginals_j[k] = 0;
    }
    for (int k = 0; k < n_i; k++) {
      for (int l = 0; l < n_j; l++) {
        int entry = contingency_matrix[k * n_j + l];
        marginals_i[k] += entry;
        marginals_j[l] += entry;
      }
    }
    if (ci_test_chi_squared_level_0(n_data, n_i, n_j, contingency_matrix,
                                    marginals_i, marginals_j)) {
      G[i * n_node + j] = 0;
      G[j * n_node + i] = 0;
    }
  }
}

__device__ void ci_test_chi_squared_level_n(double *chi_squared, int n_data,
                                            int dim_s, int n_i, int n_j,
                                            int *N_i_j_s, int *N_i_s,
                                            int *N_j_s, int *N_s,
                                            bool *result) {
  int *df = reinterpret_cast<int *>(chi_squared + 1);
  if (threadIdx.x == 0) {
    *chi_squared = 0;
    *df = 0;
  }
  __syncthreads();
  for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
    if (N_s[g] == 0) continue;
    int alx = 0, aly = 0;
    for (int l = 0; l < n_j; l++) {
      aly += (N_j_s[g * n_j + l] > 0);
    }
    for (int k = 0; k < n_i; k++) {
      if (!N_i_s[g * n_i + k]) continue;
      alx++;
      for (int l = 0; l < n_j; l++) {
        double expected = static_cast<double>(N_i_s[g * n_i + k]) *
                          N_j_s[g * n_j + l] / N_s[g];
        if (expected == 0) continue;
        double observed = N_i_j_s[g * n_i * n_j + k * n_j + l];
        double sum_term =
            (observed - expected) * (observed - expected) / expected;
        myAtomicAdd(chi_squared, sum_term);
      }
    }
    alx = (alx >= 1 ? alx : 1);
    aly = (aly >= 1 ? aly : 1);
    atomicAdd(df, (alx - 1) * (aly - 1));
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    if (*df == 0) {
      *result = true;
    } else {
      double pval = pchisq(*chi_squared, *df);
      *result = (pval >= 0.01);
    }
  }
}

__device__ void ci_test_mi_level_n(double *mi, int n_data, int dim_s, int n_i,
                                   int n_j, int *N_i_j_s, int *N_i_s,
                                   int *N_j_s, int *N_s, bool *result) {
  if (threadIdx.x == 0) {
    *mi = 0;
  }
  __syncthreads();
  for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
    if (N_s[g] == 0) continue;
    for (int k = 0; k < n_i; k++) {
      for (int l = 0; l < n_j; l++) {
        if (!N_i_j_s[g * n_i * n_j + k * n_j + l]) continue;
        double sum_term =
            static_cast<double>(N_i_j_s[g * n_i * n_j + k * n_j + l]) / n_data *
            log2(
                static_cast<double>(N_s[g]) *
                N_i_j_s[g * n_i * n_j + k * n_j + l] /
                (static_cast<double>(N_i_s[g * n_i + k]) * N_j_s[g * n_j + l]));
        myAtomicAdd(mi, sum_term);
      }
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    *result = (*mi < 0.003);
  }
}

__device__ void ci_test_sc_level_n(double *mi, int n_data, int dim_s, int n_i,
                                   int n_j, int *N_i_j_s, int *N_i_s,
                                   int *N_j_s, int *N_s, bool *result,
                                   double *regret) {
  double *r_i = mi + 1;
  double *r_ij = mi + 2;
  double *r_j = mi + 3;
  double *r_ji = mi + 4;
  if (threadIdx.x == 0) {
    *mi = *r_i = *r_ij = *r_j = *r_ji = 0;
  }
  __syncthreads();

  for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
    if (N_s[g]) {
      myAtomicAdd(r_i, log2(regret[N_s[g] * max_dim + n_i - 1]));
      myAtomicAdd(r_j, log2(regret[N_s[g] * max_dim + n_j - 1]));
    }
    for (int k = 0; k < n_i; k++) {
      myAtomicAdd(
          r_ji, log2(max(1.0, regret[N_i_s[g * n_i + k] * max_dim + n_j - 1])));
      if (k && !N_i_s[g * n_i + k]) continue;
      for (int l = 0; l < n_j; l++) {
        if (k == 0) {
          myAtomicAdd(
              r_ij,
              log2(max(1.0, regret[N_j_s[g * n_j + l] * max_dim + n_i - 1])));
        }
        if (!N_i_j_s[g * n_i * n_j + k * n_j + l]) continue;
        double sum_term =
            static_cast<double>(N_i_j_s[g * n_i * n_j + k * n_j + l]) / n_data *
            log2(
                static_cast<double>(N_s[g]) *
                N_i_j_s[g * n_i * n_j + k * n_j + l] /
                (static_cast<double>(N_i_s[g * n_i + k]) * N_j_s[g * n_j + l]));
        myAtomicAdd(mi, sum_term);
      }
    }
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    double threshold = min(*r_ij - *r_i, *r_ji - *r_j) / n_data;
    if (threshold > 0) {
      // printf("threshold: %.7lf\n", threshold);
    } else {
      // printf("r_i: %.7lf, r_ij: %.7lf, r_j: %.7lf, r_ji: %.7lf, mi: %.7lf\n",
      //        *r_i, *r_ij, *r_j, *r_ji, *mi);
    }
    *result = (*mi <= threshold);
  }
}

__device__ void ci_test_g2_level_n(double *g2, int n_data, int dim_s, int n_i,
                                   int n_j, int *N_i_j_s, int *N_i_s,
                                   int *N_j_s, int *N_s, bool *result) {
  int *df = reinterpret_cast<int *>(g2 + 1);
  if (threadIdx.x == 0) {
    *g2 = 0;
    *df = 0;
  }
  __syncthreads();
  for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
    if (N_s[g] == 0) continue;
    int alx = 0, aly = 0;
    for (int l = 0; l < n_j; l++) {
      aly += (N_j_s[g * n_j + l] > 0);
    }
    for (int k = 0; k < n_i; k++) {
      if (!N_i_s[g * n_i + k]) continue;
      alx++;
      for (int l = 0; l < n_j; l++) {
        double expected = static_cast<double>(N_i_s[g * n_i + k]) *
                          N_j_s[g * n_j + l] / N_s[g];
        if (expected == 0) continue;
        double observed = N_i_j_s[g * n_i * n_j + k * n_j + l];
        if (observed != 0) {
          double sum_term = 2 * observed * log(observed / expected);
          myAtomicAdd(g2, sum_term);
        }
      }
    }
    alx = (alx >= 1 ? alx : 1);
    aly = (aly >= 1 ? aly : 1);
    atomicAdd(df, (alx - 1) * (aly - 1));
  }
  __syncthreads();
  if (threadIdx.x == 0) {
    if (*df == 0) {
      *result = true;
    } else {
      double pval = pchisq(*g2, *df);
      *result = (pval >= 0.05);
    }
  }
}

__device__ void ci_test_bayes_factor_level_n(double *scratch_ptr, int n_data,
                                             int dim_s, int n_i, int n_j,
                                             int *N_i_j_s, int *N_i_s,
                                             int *N_j_s, int *N_s,
                                             bool *result) {
  if (threadIdx.x == 0) {
    *scratch_ptr = 0;
  }
  __syncthreads();
  const double alpha = 0.5;
  // independent score
  for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
    if (N_s[g] == 0) continue;
    double sum_term = 0;
    for (int k = 0; k < n_i; k++) {
      sum_term += lgamma(N_i_s[g * n_i + k] + alpha) - lgamma(alpha);
    }
    sum_term += lgamma(n_i * alpha) - lgamma(n_i * alpha + N_s[g]);
    for (int l = 0; l < n_j; l++) {
      sum_term += lgamma(N_j_s[g * n_j + l] + alpha) - lgamma(alpha);
    }
    sum_term += lgamma(n_j * alpha) - lgamma(n_j * alpha + N_s[g]);
    myAtomicAdd(scratch_ptr, sum_term);
  }
  __syncthreads();
  double independent_score = *scratch_ptr;
  // dependent score
  if (threadIdx.x == 0) {
    *scratch_ptr = 0;
  }
  __syncthreads();
  for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
    if (N_s[g] == 0) continue;
    double sum_term = 0;
    for (int k = 0; k < n_i; k++) {
      for (int l = 0; l < n_j; l++) {
        sum_term += lgamma(N_i_j_s[g * n_i * n_j + k * n_j + l] + alpha) -
                    lgamma(alpha);
      }
    }
    sum_term += lgamma(n_i * n_j * alpha) - lgamma(n_i * n_j * alpha + N_s[g]);
    myAtomicAdd(scratch_ptr, sum_term);
  }
  __syncthreads();
  double dependent_score = *scratch_ptr;
  if (threadIdx.x == 0) {
    // printf("independent_score: %.7lf, dependent_score: %.7lf\n",
    //        independent_score, dependent_score);
    *result = (independent_score > dependent_score - 1e-10);
  }
}

__global__ void PC_level_n(int level, int n_node, int n_data, uint8_t *data,
                           int *G, int *n_states, bool use_working_memory,
                           int *working_memory, int *sepsets, double *regret,
                           int *model, int *stats) {
  extern __shared__ int smem[];
  for (int i = blockIdx.x; i < n_node; i += gridDim.x) {
    for (int idx_j = blockIdx.y; idx_j < n_node; idx_j += gridDim.y) {
      __syncthreads();
      int *G_compacted = smem;
      if (threadIdx.x == 0 && threadIdx.y == 0) {
        int cnt = 0;
        for (int j = 0; j < n_node; j++) {
          if (G[i * n_node + j]) {
            G_compacted[++cnt] = j;
          }
        }
        G_compacted[0] = cnt;
      }
      __syncthreads();
      int n_adj = G_compacted[0];
      if (idx_j >= n_adj || n_adj - 1 < level) {
        break;
      }
      int max_dim_s = pow(static_cast<double>(max_dim), level);
      int reserved_size_per_ci_test =
          max_dim_s * max_dim * max_dim + 2 * max_dim_s * max_dim + max_dim_s;
      int j = G_compacted[idx_j + 1];
      int n_i = n_states[i];
      int n_j = n_states[j];
      int sepset_cnt = binom(n_adj - 1, level);
      int ci_test_idx = threadIdx.y;
      int *thread_memory;
      if (use_working_memory) {
        int thread_memory_index = (gridDim.y * blockDim.y) * blockIdx.x +
                                  blockDim.y * blockIdx.y + ci_test_idx;
        thread_memory =
            working_memory + thread_memory_index * reserved_size_per_ci_test;
      } else {
        thread_memory = smem + n_adj + 2 + level * blockDim.y +
                        reserved_size_per_ci_test * ci_test_idx;
      }
      int *sepset = smem + n_adj + 2 + level * ci_test_idx;
      int sepset_cnt_loop =
          (sepset_cnt + blockDim.y - 1) / blockDim.y * blockDim.y;
      for (int sepset_idx = threadIdx.y; sepset_idx < sepset_cnt_loop;
           sepset_idx += blockDim.y) {
        __syncthreads();
        int *valid = smem + n_adj + 1;
        if (threadIdx.x == 0) {
          *valid = (G[i * n_node + j] == 1);
        }
        __syncthreads();
        if (*valid == 0) break;
        if (threadIdx.x == 0) {
          comb(n_adj - 1, level, sepset_idx, idx_j, sepset);
          for (int k = 0; k < level; k++) {
            sepset[k] = G_compacted[sepset[k] + 1];
          }
          uint smid;
          asm volatile("mov.u32 %0, %smid;" : "=r"(smid));
          stats[smid]++;
        }
        __syncthreads();
        int dim_s = 1;
        for (int k = 0; k < level; k++) {
          dim_s *= n_states[sepset[k]];
        }
        int *N_i_j_s = thread_memory;
        int *N_i_s = N_i_j_s + dim_s * n_i * n_j;
        int *N_j_s = N_i_s + dim_s * n_i;
        int *N_s = N_j_s + dim_s * n_j;
        if (threadIdx.x == 0) {
          int malloc_size =
              dim_s * n_i * n_j + dim_s * n_i + dim_s * n_j + dim_s;
          memset(N_i_j_s, 0, malloc_size * sizeof(int));
        }
        __syncthreads();
        for (int k = threadIdx.x; k < n_data; k += blockDim.x) {
          int val_i = data[i * n_data + k];
          int val_j = data[j * n_data + k];
          int sepset_idx = 0;
          for (int l = 0; l < level; l++) {
            sepset_idx =
                sepset_idx * n_states[sepset[l]] + data[sepset[l] * n_data + k];
          }
          atomicAdd(N_i_j_s + sepset_idx * n_i * n_j + val_i * n_j + val_j, 1);
        }
        __syncthreads();
        for (int g = threadIdx.x; g < dim_s; g += blockDim.x) {
          for (int k = 0; k < n_i; k++) {
            for (int l = 0; l < n_j; l++) {
              int entry = N_i_j_s[g * n_i * n_j + k * n_j + l];
              atomicAdd(N_i_s + g * n_i + k, entry);
              atomicAdd(N_j_s + g * n_j + l, entry);
              atomicAdd(N_s + g, entry);
            }
          }
        }
        int scratch_addr =
            n_adj + 2 +
            (level + (use_working_memory ? 0 : reserved_size_per_ci_test)) *
                blockDim.y;
        scratch_addr = (scratch_addr + 1) / 2 * 2;
        double *scratch_ptr =
            reinterpret_cast<double *>(smem + scratch_addr) + ci_test_idx * 5;
        bool result;
        ci_test_chi_squared_level_n(scratch_ptr, n_data, dim_s, n_i, n_j,
                                    N_i_j_s, N_i_s, N_j_s, N_s, &result);
        if (threadIdx.x == 0 && result) {
          int ij_min = (i < j ? i : j);
          int ij_max = (i < j ? j : i);
          if (atomicCAS(G + ij_min * n_node + ij_max, 1, -1) == 1) {
            G[ij_max * n_node + ij_min] = -1;
            for (int k = 0; k < level; k++) {
              sepsets[(ij_min * n_node + ij_max) * max_level + k] = sepset[k];
            }
          }
        }
        __syncthreads();
      }
    }
  }
}

PDAG PCsearch(int n_node, int n_data, const vector<uint8_t> &data,
              const vector<int> &n_states, const vector<int> &model) {
  vector<int> G(n_node * n_node);
  for (int i = 0; i < n_node; i++) {
    for (int j = 0; j < n_node; j++) {
      if (i != j) G[i * n_node + j] = 1;
    }
  }
  vector<int> sepsets(n_node * n_node * max_level, -1);
  uint8_t *data_d;
  vector<double> regret(n_data * max_dim * 2);
  vector<int> stats(128);
  int *G_d, *n_states_d, *working_memory_d, *sepsets_d, *model_d, *stats_d;
  double *regret_d;
  int size_G = sizeof(int) * n_node * n_node;
  int size_data = sizeof(uint8_t) * n_data * n_node;
  int size_n_states = sizeof(int) * n_node;
  int size_working_memory = sizeof(int) * 500'000'000;
  int size_sepsets = sizeof(int) * n_node * n_node * max_level;
  int size_regret = sizeof(double) * n_data * max_dim * 2;
  int size_model = sizeof(int) * n_node * n_node * 2;
  int size_stats = sizeof(int) * 128;
  CUDA_CHECK(hipMalloc(&G_d, size_G));
  CUDA_CHECK(hipMalloc(&data_d, size_data));
  CUDA_CHECK(hipMalloc(&n_states_d, size_n_states));
  CUDA_CHECK(hipMalloc(&working_memory_d, size_working_memory));
  CUDA_CHECK(hipMalloc(&sepsets_d, size_sepsets));
  CUDA_CHECK(hipMalloc(&regret_d, size_regret));
  CUDA_CHECK(hipMalloc(&model_d, size_model));
  CUDA_CHECK(hipMalloc(&stats_d, size_stats));
  CUDA_CHECK(
      hipMemcpy(data_d, data.data(), size_data, hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(n_states_d, n_states.data(), size_n_states,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(hipMemcpy(sepsets_d, sepsets.data(), size_sepsets,
                        hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(model_d, model.data(), size_model, hipMemcpyHostToDevice));
  CUDA_CHECK(
      hipMemcpy(stats_d, stats.data(), size_stats, hipMemcpyHostToDevice));

  calc_regret<<<n_data / 1024 + 1, 1024>>>(regret_d);
  CUDA_CHECK(
      hipMemcpy(regret.data(), regret_d, size_regret, hipMemcpyDeviceToHost));
  cout << "---regret---" << endl;
  for (int n = n_data - 10; n <= n_data; n++) {
    for (int k = 1; k <= max_dim; k++) {
      cout << regret[n * max_dim + k - 1] << " ";
    }
    cout << endl;
  }
  cout << "---" << endl;
  // stage 1: Do CI tests between nodes and remove edge (undirected graph)
  int level = 0;
  int max_n_adj = n_node - 1;
  uint64_t max_dim_s = 1;
  while (level <= n_node - 2) {
    CUDA_CHECK(hipMemcpy(G_d, G.data(), size_G, hipMemcpyHostToDevice));
    stats = vector<int>(128);
    CUDA_CHECK(
        hipMemcpy(stats_d, stats.data(), size_stats, hipMemcpyHostToDevice));
    cout << "level: " << level << ", max_n_adj: " << max_n_adj << endl;
    if (level > max_level) break;
    if (level == 0) {
      dim3 threadsPerBlock(64);
      dim3 numBlocks(n_node, n_node);
      PC_level_0<<<numBlocks, threadsPerBlock>>>(
          n_node, n_data, data_d, G_d, n_states_d, regret_d, model_d, stats_d);
    } else {
      dim3 threadsPerBlock(64, 2);
      dim3 numBlocks(n_node, max_n_adj);
      uint64_t reserved_size_per_ci_test =
          max_dim_s * max_dim * max_dim + 2 * max_dim_s * max_dim + max_dim_s;
      if (reserved_size_per_ci_test * 2 > size_working_memory / sizeof(int)) {
        cout << "working memory is not enough" << endl;
        break;
      }
      if (reserved_size_per_ci_test * 2 < 1000) {
        PC_level_n<<<numBlocks, threadsPerBlock,
                     sizeof(int) * (max_n_adj + 2 +
                                    (level + reserved_size_per_ci_test) * 2) +
                         sizeof(double) * (10 + 1)>>>(
            level, n_node, n_data, data_d, G_d, n_states_d, false, nullptr,
            sepsets_d, regret_d, model_d, stats_d);
      } else {
        uint64_t reserved_size_per_row =
            reserved_size_per_ci_test * 2 * max_n_adj;
        int max_rows =
            size_working_memory / sizeof(int) / reserved_size_per_row;
        if (max_rows == 0) {
          int max_columns =
              size_working_memory / sizeof(int) / 2 / reserved_size_per_ci_test;
          numBlocks.x = 1;
          numBlocks.y = max_columns;
        } else if (numBlocks.x > max_rows) {
          numBlocks.x = max_rows;
        }
        cout << "numBlocks: " << numBlocks.x << ", " << numBlocks.y << endl;
        PC_level_n<<<numBlocks, threadsPerBlock,
                     sizeof(int) * (max_n_adj + 2 + level * 2) +
                         sizeof(double) * (10 + 1)>>>(
            level, n_node, n_data, data_d, G_d, n_states_d, true,
            working_memory_d, sepsets_d, regret_d, model_d, stats_d);
      }
    }
    CUDA_CHECK(
        hipMemcpy(stats.data(), stats_d, size_stats, hipMemcpyDeviceToHost));
    // cout << "stats: " << stats[0] << ", " << stats[1] << ", " << stats[2]
    //      << ", " << stats[3] << endl;
    cout << "----stats: " << endl;
    for (int i = 0; i < 128; i++) {
      cout << i << ' ' << stats[i] << endl;
    }
    cout << "-----" << endl;
    CUDA_CHECK(hipMemcpy(G.data(), G_d, size_G, hipMemcpyDeviceToHost));
    max_n_adj = 0;
    int next_node_cnt = 0;
    int next_edge_cnt = 0;
    for (int i = 0; i < n_node; i++) {
      int n_adj = 0;
      for (int j = 0; j < n_node; j++) {
        if (G[i * n_node + j] == -1) {
          G[i * n_node + j] = 0;
        }
        if (G[i * n_node + j]) n_adj++;
      }
      if (n_adj - 1 > level) {
        next_node_cnt++;
      }
      max_n_adj = max(max_n_adj, n_adj);
      next_edge_cnt += n_adj;
    }
    cout << "next_node_cnt, next_edge_cnt: " << next_node_cnt << ", "
         << next_edge_cnt / 2 << endl;
    if (max_n_adj - 1 <= level) break;
    level++;
    max_dim_s *= max_dim;
  }
  CUDA_CHECK(hipMemcpy(sepsets.data(), sepsets_d, size_sepsets,
                        hipMemcpyDeviceToHost));
  CUDA_CHECK(hipFree(G_d));
  CUDA_CHECK(hipFree(data_d));
  CUDA_CHECK(hipFree(n_states_d));
  CUDA_CHECK(hipFree(working_memory_d));
  CUDA_CHECK(hipFree(sepsets_d));
  // stage 2: orient edges
  PDAG G_pdag;
  G_pdag.g = vector<vector<bool>>(n_node, vector<bool>(n_node));
  for (int i = 0; i < n_node; i++) {
    for (int j = 0; j < n_node; j++) {
      G_pdag.g.at(i).at(j) = G[i * n_node + j];
    }
  }
  orientation(G_pdag, sepsets);
  return G_pdag;
}

py::array_t<bool> gpuPC(py::array_t<uint8_t> data, py::array_t<int> n_states,
                        py::array_t<bool> true_model) {
  // translate input data to c++ vector(this is not optimal but I don't know
  // how to use pybind11::array_t)
  py::buffer_info buf_data = data.request(), buf_states = n_states.request(),
                  buf_model = true_model.request();
  const uint8_t *__restrict__ prt_data = static_cast<uint8_t *>(buf_data.ptr);
  const int *__restrict__ prt_states = static_cast<int *>(buf_states.ptr);
  const bool *__restrict__ prt_model = static_cast<bool *>(buf_model.ptr);
  size_t n_data = buf_data.shape[0],
         n_node = buf_data.shape[1];  // number of nodes
  cout << "n_data, n_node: " << n_data << ' ' << n_node << endl;
  vector<uint8_t> data_vec(n_data * n_node);
  vector<int> n_states_vec(n_node);
  vector<int> model_vec(2 * n_node * n_node);
  for (size_t i = 0; i < n_data; i++) {
    for (size_t j = 0; j < n_node; j++) {
      data_vec.at(j * n_data + i) = prt_data[i * n_node + j];
    }
  }
  for (size_t i = 0; i < n_node; i++) {
    n_states_vec.at(i) = prt_states[i];
  }
  for (size_t i = 0; i < n_node; i++) {
    int cnt = 0;
    for (size_t j = 0; j < n_node; j++) {
      if (prt_model[i * n_node + j]) {
        model_vec.at(i * n_node + (++cnt)) = j;
      }
    }
    model_vec.at(i * n_node) = cnt;
  }
  for (size_t i = 0; i < n_node; i++) {
    int cnt = 0;
    for (size_t j = 0; j < n_node; j++) {
      if (prt_model[j * n_node + i]) {
        model_vec.at(n_node * n_node + i * n_node + (++cnt)) = j;
      }
    }
    model_vec.at(n_node * n_node + i * n_node) = cnt;
  }
  auto endg = py::array_t<bool>({n_node, n_node});
  py::buffer_info buf_endg = endg.request();
  bool *__restrict__ prt_endg = static_cast<bool *>(buf_endg.ptr);

  PDAG Gend = PCsearch(n_node, n_data, data_vec, n_states_vec, model_vec);

  // translate Gend to py::array_t (this is not optimal but I don't know how
  // to use pybind11::array_t)
  for (size_t i = 0; i < n_node; i++) {
    for (size_t j = 0; j < n_node; j++) {
      prt_endg[i * n_node + j] = Gend.g.at(i).at(j);
    }
  }
  return endg;
}
}  // namespace cuda